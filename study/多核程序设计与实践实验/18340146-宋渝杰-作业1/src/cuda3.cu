
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

int sizex = 32, sizey = 32;

struct Matrix{
	int x, y;
	int** v;
}; 

void init(Matrix* m) {
	for (int i = 0; i < m->x; i++)
		for (int j = 0; j < m->y; j++)
			m->v[i][j] = rand() % 10;
}

void print(Matrix* m) {
	for (int i = 0; i < m->x; i++) {
		for (int j = 0; j < m->y; j++) printf("%d ", m->v[i][j]);
		printf("\n");
	}
}

__global__ void add(Matrix* a, Matrix* b, Matrix* c) {
	int x = blockIdx.y * blockDim.y + threadIdx.y, y = blockIdx.x * blockDim.x + threadIdx.x;
	if (x < c->x && y < c->y)
		c->v[x][y] = a->v[x][y] + b->v[x][y];
}

int main(int argc, char* argv[]) {
	int m = strtol(argv[1], NULL, 10), n = strtol(argv[2], NULL, 10);
	timeval t1, t2;
	Matrix *a, *b, *c;
	hipMallocManaged((void**)&a, sizeof(Matrix));
	hipMallocManaged((void**)&b, sizeof(Matrix));
	hipMallocManaged((void**)&c, sizeof(Matrix));
	a->x = m; a->y = n; b->x = m; b->y = n; c->x = m; c->y = n;
	hipMallocManaged((void**)&a->v, a->x * sizeof(int*));
	hipMallocManaged((void**)&b->v, b->x * sizeof(int*));
	hipMallocManaged((void**)&c->v, c->x * sizeof(int*));
    for (int i = 0; i < m; i++) {
        hipMallocManaged((void**)&a->v[i], a->y * sizeof(int));
        hipMallocManaged((void**)&b->v[i], b->y * sizeof(int));
        hipMallocManaged((void**)&c->v[i], c->y * sizeof(int));
    }
	init(a); init(b);
	dim3 block(sizex, sizey), grid(m / sizex, n / sizey);
	
    gettimeofday(&t1, NULL);
	add <<< grid, block >>> (a, b, c);
    hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    
    printf("Time: %.4fs\n", (t2.tv_sec-t1.tv_sec+(t2.tv_usec-t1.tv_usec)/1.0e6));
    /*
    printf("Matrix a: \n");
    print(a);
    printf("Matrix b: \n");
    print(b);
    printf("Matrix c: \n");
    print(c);
    */
    return 0;
}