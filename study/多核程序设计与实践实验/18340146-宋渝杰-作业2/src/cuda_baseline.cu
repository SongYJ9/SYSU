/*
#############################################################
##  姓名：宋渝杰
##  文件说明：cuda实验二的baseline版本程序
#############################################################
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

int sizex = 8, sizey = 8;                       // blocksize
int readInFlie = true, printMatrix = false;     // 是否从文件读入输入矩阵、是否输出矩阵

struct Matrix { // 矩阵结构体
    int x, y;
    double** v;
}; 

/*
#############################################################
##  函数：print
##  函数描述：按二维矩阵形式输出矩阵
##  参数描述：
##  Matrix *m：Matrix类型的结构体矩阵
#############################################################
*/

void print(Matrix* m) {
    for (int i = 0; i < m->x; i++) {
        for (int j = 0; j < m->y; j++) printf("%12.5f ", m->v[i][j]);
        printf("\n");
    }
}

/*
#############################################################
##  函数：cal
##  函数描述：核函数，计算输入矩阵a的熵，计算结果存在矩阵b里
##  参数描述：
##  Matrix *a：输入矩阵a
##  Matrix *b：结果矩阵b
#############################################################
*/

__global__ void cal(Matrix* a, Matrix* b) {
    int x = blockIdx.y * blockDim.y + threadIdx.y, y = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < a->x && y < a->y) {
        int num[16] = {0}, sum = 0;
        double ans = 0;
        for (int i = max(x - 2, 0); i < min(x + 3, a->x); i++) {
            for (int j = max(y - 2, 0); j < min(y + 3, a->y); j++) {
                num[(int)(a->v[i][j])]++;
                sum++;
            }
        }
        for (int i = 0; i < 16; i++)
            if (num[i]) ans -= (double)num[i] / sum * log((double)num[i] / sum);
        b->v[x][y] = ans;
    }
}

/*
#############################################################
##  函数：main
##  函数描述：程序主函数，负责读入矩阵、申请内存、调用核函数、输出计算时间和矩阵等工作
##  参数描述：
##  int argc, char* argv[]：可变输入参数，实际上只接受第一个输入，即输入文件名
#############################################################
*/

int main(int argc, char* argv[]) {

    // 读入输入矩阵、申请内存
    if (readInFlie) freopen(argv[1], "r", stdin);
    Matrix *a, *b;
    hipMallocManaged((void**)&a, sizeof(Matrix));
    hipMallocManaged((void**)&b, sizeof(Matrix));
    scanf("%d%d", &a->x, &a->y);
    b->x = a->x; b->y = a->y;
    hipMallocManaged((void**)&a->v, a->x * sizeof(double*));
    hipMallocManaged((void**)&b->v, b->x * sizeof(double*));
    for (int i = 0; i < a->x; i++) {
        hipMallocManaged((void**)&a->v[i], a->y * sizeof(double));
        hipMallocManaged((void**)&b->v[i], b->y * sizeof(double));
    }
    for (int i = 0; i < a->x; i++)
        for (int j = 0; j < a->y; j++)
            scanf("%lf", &a->v[i][j]);
    
    // 设置线程块和网格大小
    dim3 block(sizex, sizey), grid(a->x / sizex + 1, a->y / sizey + 1);
    
    // 调用核函数并计时
    timeval t1, t2;
    gettimeofday(&t1, NULL);
    cal <<< grid, block >>> (a, b);
    hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    
    // 输出结果矩阵
    printf("Time: %.4fs\n", (t2.tv_sec-t1.tv_sec+(t2.tv_usec-t1.tv_usec)/1.0e6));
    if (printMatrix) {
        printf("Matrix a: \n");
        print(a);
        printf("\nMatrix b: \n");
        print(b); 
    }
    return 0;
}