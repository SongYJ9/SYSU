/*
#############################################################
##  姓名：宋渝杰
##  文件说明：cuda实验二的最终优化版本程序
#############################################################
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

short sizex = 256, sizey = 4;                   // blocksize
__constant__ short X, Y;                        // 常量内存存储矩阵高和宽
int readInFlie = true, printMatrix = false;     // 是否从文件读入输入矩阵、是否输出矩阵

/*
#############################################################
##  函数：print
##  函数描述：按二维矩阵形式输出矩阵（short类型和float类型）
##  参数描述：
##  T *m：类型为T的一维矩阵
##  int X：矩阵的高
##  int Y：矩阵的宽
#############################################################
*/

template<class T>
void print(T *m, int X, int Y) {
    for (int i = 0; i < X; i++) {
        for (int j = 0; j < Y; j++) printf("%12.5f ", (float)m[i * Y + j]); // 保留五位小数输出
        printf("\n");
    }
}

/*
#############################################################
##  函数：cal
##  函数描述：核函数，计算输入矩阵a的熵，计算结果存在矩阵b里
##  参数描述：
##  short *a：输入矩阵a
##  float *b：结果矩阵b
#############################################################
*/

__global__ void cal(short *a, float *b) {
    short x = blockIdx.y * blockDim.y + threadIdx.y, y = blockIdx.x * blockDim.x + threadIdx.x; // 定位线程id
    if (x < X && y < Y) {
        short num[16] = {0}, sum = 0;   // 记录0-15数值的数量、窗口实际大小
        float ans = 0;                  // 记录这个位置熵的结果
        __shared__ float loge[26];      // log共享内存打表
        loge[0] = 0.0; loge[1] = 0.0; loge[2] = 0.693147; loge[3] = 1.098612; loge[4] = 1.386294; loge[5] = 1.609437;
        loge[6] = 1.791759; loge[7] = 1.945910; loge[8] = 2.079441; loge[9] = 2.197224; loge[10] = 2.302585;
        loge[11] = 2.397895; loge[12] = 2.484906; loge[13] = 2.564949; loge[14] = 2.639057; loge[15] = 2.708050;
        loge[16] = 2.772588; loge[17] = 2.833213; loge[18] = 2.890371; loge[19] = 2.944438; loge[20] = 2.995732;
        loge[21] = 3.044522; loge[22] = 3.091042; loge[23] = 3.135494; loge[24] = 3.178053; loge[25] = 3.218875;
        for (short i = max(x - 2, 0); i < min(x + 3, X); i++) {
            for (short j = max(y - 2, 0); j < min(y + 3, Y); j++) {
                num[a[i * Y + j]]++;    // 该数值数量+1
                sum++;                  // 窗口实际大小+1
            }
        }
        for (short i = 0; i < 16; i++)  // 计算熵（按照熵的计算公式计算）
            if (num[i]) ans -= (float)num[i] / sum * (loge[num[i]] - loge[sum]);
        b[x * Y + y] = ans;
    }
}

/*
#############################################################
##  函数：main
##  函数描述：程序主函数，负责读入矩阵、申请内存、调用核函数、输出计算时间和矩阵等工作
##  参数描述：
##  int argc, char* argv[]：可变输入参数，实际上只接受第一个输入，即输入文件名
#############################################################
*/

int main(int argc, char* argv[]) {

    // 读入输入矩阵、申请内存
    if (readInFlie) freopen(argv[1], "r", stdin);
    short x, y, *a;
    float *b;
    scanf("%d%d", &x, &y);
    hipMallocManaged((void**)&a, x * y * sizeof(short));
    hipMallocManaged((void**)&b, x * y * sizeof(float));
    for (short i = 0; i < x; i++)
        for (short j = 0; j < y; j++)
            scanf("%hd", &a[i * y + j]);
    hipMemcpyToSymbol(HIP_SYMBOL(X), &x, sizeof(short));
    hipMemcpyToSymbol(HIP_SYMBOL(Y), &y, sizeof(short)); 
    
    // 设置线程块和网格大小
    dim3 block(sizex, sizey), grid(x / sizex + 1, y / sizey + 1);
    
    // 调用核函数并计时
    timeval t1, t2;
    gettimeofday(&t1, NULL);
    cal <<< grid, block >>> (a, b);
    hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    
    // 输出结果矩阵
    printf("Time: %.4fs\n", (t2.tv_sec-t1.tv_sec+(t2.tv_usec-t1.tv_usec)/1.0e6));
    if (printMatrix) {
        printf("Matrix a: \n");
        print(a, x, y);
        printf("\nMatrix b: \n");
        print(b, x, y);
    }
    return 0;
}