#include "hip/hip_runtime.h"
/*
#############################################################
##  姓名：宋渝杰、黄思蓉
##  文件说明：cuda大作业“模拟退火解决TSP问题”baseline版本程序
#############################################################
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>

short blocksize = 512;
bool readInFlie = true;

/*
#############################################################
##  函数：SA_TSP
##  函数描述：并行模拟退火算法解决TSP问题
##  参数描述：
##  int *x：城市的x坐标
##  int *y：城市的y坐标
##  int n：城市个数
##  int *path：返回每个线程的回路路径
##  double *len：返回每个线程的回路长度
##  int seed：cpu传来的随机种子
#############################################################
*/

__global__ void SA_TSP(int *x, int *y, int n, int *path, double *len, int seed) {

    // 初始化工作
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;              // 计算线程ID
    int *threadPath = new int[n], *changePath = new int[n];            // 线程计算的回路路线
    double threadLen = 1e18, changeLen = 0;                            // 线程计算的回路长度
    double T = 1e3, alpha = 0.95;                                      // 模拟退火的温度和降温率
    int L = 10;
    hiprandState state;
    hiprand_init(seed, threadId, 0, &state);                            // 核函数随机函数
    for (int i = 0; i < n; i++) changePath[i] = i;                     // 初始化为1->2->...->n->1的回路
    
    // 降温迭代过程
    while (T > 1e-3) {
        int random = hiprand(&state) % 100;                             // 随机变换
        if (random < 20) {
            int t1 = hiprand(&state) % (n - 1);
            int t2 = t1 + 1;
            int temp = changePath[t1]; 
            changePath[t1] = changePath[t2]; 
            changePath[t2] = temp;                                     // 交换相邻的两个点
        }
        else if (random < 40) {
            int t1, t2;
            do {
                t1 = hiprand(&state) % n;
                t2 = hiprand(&state) % n;
            } while (t1 >= t2);
            int temp = changePath[t1];
            changePath[t1] = changePath[t2];
            changePath[t2] = temp;                                     // 交换任意的两个点
        }
        else if (random < 70) {
            int t1, t2;
            do {
                t1 = hiprand(&state) % n;
                t2 = hiprand(&state) % n;
            } while (t1 >= t2);
            for (int i = t1, j = t2; i < j; i++, j--) {
                int temp = changePath[i];
                changePath[i] = changePath[j];
                changePath[j] = temp;                                  // 翻转[t1, t2]序列
            }
        }
        else {
            int t1, t2;
            do {
                t1 = hiprand(&state) % n;
                t2 = hiprand(&state) % (n / 3);
            } while (t1 - t2 < 0 || t1 + t2 >= n);
            for (int i = t1 - t2, j = t1; i < t1; i++, j++) {
                int temp = changePath[i];
                changePath[i] = changePath[j];
                changePath[j] = temp;                                  // 交换[t1-t2, t1)和[t1, t1+t2)序列
            }
        }
        
        // 计算变换后的回路长度
        changeLen = 0;
        for (int i = 1; i < n; i++) 
            changeLen += sqrt(pow(x[changePath[i]] - x[changePath[i-1]], 2)
                + pow(y[changePath[i]] - y[changePath[i-1]], 2));
        changeLen += sqrt(pow(x[changePath[n-1]] - x[changePath[0]], 2)
                + pow(y[changePath[n-1]] - y[changePath[0]], 2));
        
        // 如果变换后回路更优，则接受本次变换，否则以一定概率接受本次变换
        if (changeLen < threadLen || hiprand(&state) % 100 < exp(-(changeLen - threadLen)/T) * 100) {
            threadLen = changeLen;
            for (int i = 0; i < n; i++) threadPath[i] = changePath[i];
        }
        else {
            for (int i = 0; i < n; i++) changePath[i] = threadPath[i];
        }
        
        // 迭代次数足够后，进行温度下降
        if (L-- == 0) {
            L = 10;
            T *= alpha;
        }
    }
    
    // 返回线程结果
    for (int i = 0; i < n; i++) path[threadId * n + i] = threadPath[i];
    len[threadId] = threadLen;
    delete(threadPath); delete(changePath);
}

/*
#############################################################
##  函数：main
##  函数描述：主函数，负责文件输入、内存申请、调用核函数、计算最优结果并输出。
##  参数描述：
##  int argc, char* argv[]：可变输入参数，实际上只接受第一个输入，即输入文件名
#############################################################
*/

int main(int argc, char* argv[]) {

    hipSetDevice(2);
    // 读入城市位置、申请内存
    if (readInFlie) freopen(argv[1], "r", stdin);
    srand((unsigned int)time(NULL));
    int n, *x, *y, *ans;
    double *len;
    scanf("%d", &n);
    hipMallocManaged((void**)&x, n * sizeof(int));
    hipMallocManaged((void**)&y, n * sizeof(int));
    hipMallocManaged((void**)&ans, blocksize * n * sizeof(int));
    hipMallocManaged((void**)&len, blocksize * sizeof(double));
    for (int i = 0; i < n; i++) scanf("%d%d", &x[i], &y[i]);

    // 设置线程块和网格大小
    dim3 block(blocksize), grid(1);
    
    // 调用核函数并计时
    timeval t1, t2;
    gettimeofday(&t1, NULL);
    SA_TSP <<< grid, block >>> (x, y, n, ans, len, rand());
    hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
    
    // 计算最优结果
    double mi = 1e18; int index = -1;
    for (int i = 0; i < blocksize; i++) {
        if (len[i] < mi) {
            mi = len[i];
            index = i;
        }
    }
    
    // 输出最优结果
    printf("Time: %.4fs\n", (t2.tv_sec-t1.tv_sec+(t2.tv_usec-t1.tv_usec)/1.0e6));
    printf("Final result: %d \nPath: ", index);
    for (int i = 0; i < n; i++) printf("%d -> ", ans[index * n + i]);
    printf("%d\n", ans[index * n]);
    printf("Length: %8.2f\n\n", len[index]);
    
    return 0;
}