/*
#############################################################
##  姓名：宋渝杰、黄思蓉
##  文件说明：cuda大作业“模拟退火解决TSP问题”final版本程序
#############################################################
*/

#include "TSP_final.h"

/*
#############################################################
##  函数：main
##  函数描述：主函数，负责文件输入、内存申请、调用核函数、输出最优结果。
##  参数描述：
##  int argc, char* argv[]：可变输入参数，实际上只接受第一个输入，即输入文件名
#############################################################
*/

int main(int argc, char* argv[]) {

    // 读入城市位置
    if (readInFlie) freopen(argv[1], "r", stdin);
    srand((unsigned int)time(NULL));
    int n, *x, *y;
    scanf("%d", &n);
    x = new int[n]; y = new int[n];
    for (int i = 0; i < n; i++) scanf("%d%d", &x[i], &y[i]);
    
    // 申请cuda内存
    int GPU_num;
    hipGetDeviceCount(&GPU_num);
    int *ans[GPU_num], *index[GPU_num], *d_x[GPU_num], *d_y[GPU_num], *d_ans[GPU_num], *d_index[GPU_num];
    double *len[GPU_num], *d_len[GPU_num];
    hipStream_t stream[GPU_num];
    for (int i = 0; i < GPU_num; i++) {
        hipSetDevice(i);
        ans[i] = new int[n * gridSize];
        index[i] = new int[gridSize];
        len[i] = new double[gridSize];
        hipMalloc((void**)&d_x[i], n * sizeof(int));
        hipMalloc((void**)&d_y[i], n * sizeof(int));
        hipMalloc((void**)&d_ans[i], n * gridSize * sizeof(int));
        hipMalloc((void**)&d_index[i], blockSize * gridSize * sizeof(int));
        hipMalloc((void**)&d_len[i], blockSize * gridSize * sizeof(double));
        hipStreamCreate(&stream[i]);
    }

    // 设置线程块和网格大小
    dim3 block(blockSize), grid(gridSize);
    
    // openmp+cuda 调用核函数并计时
    timeval t1, t2;
    gettimeofday(&t1, NULL);
    #pragma omp parallel for num_threads(GPU_num)
    for (int i = 0; i < GPU_num; i++) {
        hipSetDevice(i);
        hipMemcpyAsync(d_x[i], x, n * sizeof(int), hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(d_y[i], y, n * sizeof(int), hipMemcpyHostToDevice, stream[i]);
        SA_TSP <<< grid, block, 0, stream[i]>>> (d_x[i], d_y[i], n, d_ans[i], d_len[i], d_index[i], rand());
        hipMemcpyAsync(ans[i], d_ans[i], n * gridSize * sizeof(int), hipMemcpyDeviceToHost, stream[i]);
        hipMemcpyAsync(len[i], d_len[i], gridSize * sizeof(double), hipMemcpyDeviceToHost, stream[i]);
        hipMemcpyAsync(index[i], d_index[i], gridSize * sizeof(int), hipMemcpyDeviceToHost, stream[i]);
    }
    gettimeofday(&t2, NULL);
    
    // cuda内存释放
    for (int i = 0; i < GPU_num; i++) {
        hipSetDevice(i);
        hipStreamSynchronize(stream[i]);
        hipFree(d_x[i]); hipFree(d_y[i]); 
        hipFree(d_ans[i]); hipFree(d_len[i]); hipFree(d_index[i]);
        hipStreamDestroy(stream[i]);
    }
    
    // 计算最优结果
    double mi = 1e18;
    int GPUIndex = -1, gridIndex = -1;
    for (int i = 0; i < GPU_num; i++) {
        for (int j = 0; j < gridSize; j++) {
            if (len[i][j] < mi) {
                mi = len[i][j];
                GPUIndex = i; gridIndex = j;
            }
        }
    }
    
    // 输出最优结果
    printf("Time: %.4fs\n", (t2.tv_sec-t1.tv_sec+(t2.tv_usec-t1.tv_usec)/1.0e6));
    printf("Final result: \n  GPU_index: %d\n  thread_index: %d \n  Path: ", GPUIndex, index[GPUIndex][gridIndex]);
    for (int i = 0; i < n; i++) printf("%d -> ", ans[GPUIndex][gridIndex * n + i]);
    printf("%d\n", ans[GPUIndex][gridIndex * n]);
    printf("  Length: %8.2f\n\n", len[GPUIndex][gridIndex]);
    
    // 文件写入
    freopen("../out/out.txt", "w", stdout);
    for (int i = 0; i < n; i++) printf("%d ", ans[GPUIndex][gridIndex * n + i]);
    
    // host内存释放
    delete[] x; delete[] y; 
    for (int i = 0; i < GPU_num; i++) {
        delete[] ans[i]; 
        delete[] len[i]; 
        delete[] index[i]; 
    }
    
    return 0;
}