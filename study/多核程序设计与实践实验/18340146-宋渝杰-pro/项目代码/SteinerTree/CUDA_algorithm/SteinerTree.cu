/*
#############################################################
##  姓名：宋渝杰、黄思蓉
##  文件说明：cuda大作业“模拟退火解决最小斯坦纳树问题”CUDA程序
#############################################################
*/

#include "SteinerTree.h"

/*
#############################################################
##  函数：main
##  函数描述：主函数，负责文件输入、内存申请、调用核函数、输出最优结果。
##  参数描述：
##  int argc, char* argv[]：可变输入参数，实际上只接受第一个输入，即输入文件名
#############################################################
*/

int main(int argc, char* argv[]) {

    // 读入无向连通图
    if (readInFlie) freopen(argv[1], "r", stdin);
    srand((unsigned int)time(NULL));
    int n, m, k, x, y;
    scanf("%d%d%d", &n, &m, &k);
    int *map = new int[n * n], *key = new int[k];
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++) 
            map[i * n + j] = (i == j ? 0 : 1e6);
    for (int i = 0; i < m; i++) {
        scanf("%d%d", &x, &y);
        map[x * n + y] = map[y * n + x] = 1;
    }
    for (int i = 0; i < k; i++) scanf("%d", &key[i]);
    
    // floyd算法预处理多源最短路径
    for (int l = 0; l < n; l++)
        for (int i = 0; i < n; i++)
            for (int j = 0; j < n; j++)
                if (map[i * n + j] > map[i * n + l] + map[l * n + j])
                    map[i * n + j] = map[i * n + l] + map[l * n + j];
                    
    // 申请cuda内存
    int GPU_num;
    hipGetDeviceCount(&GPU_num);
    int *d_map[GPU_num], *d_key[GPU_num];
    int *ans[GPU_num], *num[GPU_num], *len[GPU_num], *index[GPU_num];
    int *d_ans[GPU_num], *d_num[GPU_num], *d_len[GPU_num], *d_index[GPU_num];
    hipStream_t stream[GPU_num];
    for (int i = 0; i < GPU_num; i++) {
        hipSetDevice(i);
        ans[i] = new int[n * gridSize];
        num[i] = new int[gridSize];
        len[i] = new int[gridSize];
        index[i] = new int[gridSize];
        hipMalloc((void**)&d_map[i], n * n * sizeof(int));
        hipMalloc((void**)&d_key[i], k * sizeof(int));
        hipMalloc((void**)&d_ans[i], n * gridSize * sizeof(int));
        hipMalloc((void**)&d_num[i], blockSize * gridSize * sizeof(int));
        hipMalloc((void**)&d_len[i], blockSize * gridSize * sizeof(int));
        hipMalloc((void**)&d_index[i], blockSize * gridSize * sizeof(int));
        hipStreamCreate(&stream[i]);
    }

    // 设置线程块和网格大小
    dim3 block(blockSize), grid(gridSize);
    
    // openmp+cuda 调用核函数并计时
    timeval t1, t2;
    gettimeofday(&t1, NULL);
    
    #pragma omp parallel for num_threads(GPU_num)
    for (int i = 0; i < GPU_num; i++) {
        hipSetDevice(i);
        hipMemcpyAsync(d_map[i], map, n * n * sizeof(int), hipMemcpyHostToDevice, stream[i]);
        hipMemcpyAsync(d_key[i], key, k * sizeof(int), hipMemcpyHostToDevice, stream[i]);
        SA_SteinerTree <<< grid, block, 0, stream[i]>>> (d_map[i], d_key[i], n, k, d_ans[i], d_num[i], d_len[i], d_index[i], rand());
        hipMemcpyAsync(ans[i], d_ans[i], n * gridSize * sizeof(int), hipMemcpyDeviceToHost, stream[i]);
        hipMemcpyAsync(num[i], d_num[i], gridSize * sizeof(int), hipMemcpyDeviceToHost, stream[i]);
        hipMemcpyAsync(len[i], d_len[i], gridSize * sizeof(int), hipMemcpyDeviceToHost, stream[i]);
        hipMemcpyAsync(index[i], d_index[i], gridSize * sizeof(int), hipMemcpyDeviceToHost, stream[i]);
    }
    gettimeofday(&t2, NULL);
    
    // cuda内存释放
    for (int i = 0; i < GPU_num; i++) {
        hipSetDevice(i);
        hipStreamSynchronize(stream[i]);
        hipFree(d_map[i]); hipFree(d_key[i]);
        hipFree(d_ans[i]); hipFree(d_num[i]); hipFree(d_len[i]); hipFree(d_index[i]);
        hipStreamDestroy(stream[i]);
    }
    
    // 计算最优结果
    int mi = 1e6, c = 0, GPUIndex = -1, gridIndex = -1;
    for (int i = 0; i < GPU_num; i++) {
        for (int j = 0; j < gridSize; j++) {
            if (len[i][j] < mi || (len[i][j] == mi && num[i][j] > c)) {
                mi = len[i][j];
                c = num[i][j];
                GPUIndex = i; gridIndex = j;
            }
        }
    }
    
    // 输出最优结果
    printf("Time: %.4fs\n", (t2.tv_sec-t1.tv_sec+(t2.tv_usec-t1.tv_usec)/1.0e6));
    printf("Final result: \n  GPU_index: %d  thread_index: %d \n  TreeNodes: ", GPUIndex, index[GPUIndex][gridIndex]);
    int lis[num[GPUIndex][gridIndex]];
    for (int i = 0; i < num[GPUIndex][gridIndex]; i++) lis[i] = ans[GPUIndex][gridIndex*n+i];
    std::sort(lis, lis+num[GPUIndex][gridIndex]);
    for (int i = 0; i < num[GPUIndex][gridIndex]; i++) printf("%d ", lis[i]);
    printf("\n  Node_num: %d  Length: %d\n\n", num[GPUIndex][gridIndex], len[GPUIndex][gridIndex]);
    
    // 文件写入
    freopen("../out/out.txt", "w", stdout);
    for (int i = 0; i < num[GPUIndex][gridIndex]; i++) printf("%d ", lis[i]);
    
    // host内存释放
    delete[] map; delete[] key;
    for (int i = 0; i < GPU_num; i++) {
        delete[] ans[i];
        delete[] num[i];
        delete[] len[i];
        delete[] index[i];
    }
    
    return 0;
}
